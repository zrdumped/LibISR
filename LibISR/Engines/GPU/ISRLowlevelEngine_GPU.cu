#include "hip/hip_runtime.h"
// Copyright 2014-2015 Isis Innovation Limited and the authors of LibISR

#include "ISRLowlevelEngine_GPU.h"
#include "../shared/ISRLowlevelEngine_shared.h"

#include "../../Utils/IOUtil.h"

using namespace LibISR;
using namespace LibISR::Engine;
using namespace LibISR::Objects;

__global__ void subsampleImageRGBDImage_device(const Vector4f *imageData_in, Vector2i oldDims, Vector4f *imageData_out, Vector2i newDims);

__global__ void prepareAlignedRGBDData_device(Vector4f* rgbd_out, const short *depth_in, const Vector4u *rgb_in, const Vector2i imgSize, Matrix3f H, Vector3f T);

__global__ void preparePointCloudFromAlignedRGBDImage_device(Vector4f* ptcloud_out, Vector4f* inimg, float* histogram, Vector4f intrinsic, Vector4i boundingbox, Vector2i imgSize, int histBins);

__global__ void computepfImageFromHistogram_device(Vector4u* inimg, float* histogram, Vector2i imgSize, int histBins);

//////////////////////////////////////////////////////////////////////////
// host functions
//////////////////////////////////////////////////////////////////////////

void LibISR::Engine::ISRLowlevelEngine_GPU::subsampleImageRGBDImage(Float4Image *outimg, Float4Image *inimg)
{
	Vector2i oldDims = inimg->noDims;
	Vector2i newDims; newDims.x = inimg->noDims.x / 2; newDims.y = inimg->noDims.y / 2;
	outimg->ChangeDims(newDims);

	const Vector4f *imageData_in = inimg->GetData(MEMORYDEVICE_CUDA);
	Vector4f *imageData_out = outimg->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)newDims.x / (float)blockSize.x), (int)ceil((float)newDims.y / (float)blockSize.y));

	subsampleImageRGBDImage_device << <gridSize, blockSize >> >(imageData_in, oldDims, imageData_out, newDims);
}

void LibISR::Engine::ISRLowlevelEngine_GPU::prepareAlignedRGBDData(Float4Image *outimg, ShortImage *raw_depth_in, UChar4Image *rgb_in, Objects::ISRExHomography *home)
{
	int w = raw_depth_in->noDims.width;
	int h = raw_depth_in->noDims.height;

	short* depth_in_ptr = raw_depth_in->GetData(MEMORYDEVICE_CUDA);
	Vector4u* rgb_in_ptr = rgb_in->GetData(MEMORYDEVICE_CUDA);
	Vector4f* rgbd_out_ptr = outimg->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)w / (float)blockSize.x), (int)ceil((float)h / (float)blockSize.y));

	prepareAlignedRGBDData_device << <gridSize, blockSize >> >(rgbd_out_ptr, depth_in_ptr, rgb_in_ptr, rgb_in->noDims, home->H, home->T);
}

void LibISR::Engine::ISRLowlevelEngine_GPU::preparePointCloudFromAlignedRGBDImage(Float4Image *ptcloud_out, Float4Image *inimg, Objects::ISRHistogram *histogram, const Vector4f &intrinsic, const Vector4i &boundingbox)
{
	if (inimg->noDims != ptcloud_out->noDims) ptcloud_out->ChangeDims(inimg->noDims);
	
	int w = inimg->noDims.width;
	int h = inimg->noDims.height;

	int noBins = histogram->noBins;

	Vector4f *inimg_ptr = inimg->GetData(MEMORYDEVICE_CUDA);
	Vector4f* ptcloud_ptr = ptcloud_out->GetData(MEMORYDEVICE_CUDA);
	float* histogram_ptr = histogram->getPosteriorHistogram(true);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)w / (float)blockSize.x), (int)ceil((float)h / (float)blockSize.y));

	preparePointCloudFromAlignedRGBDImage_device << <gridSize, blockSize >> >(ptcloud_ptr, inimg_ptr, histogram_ptr, intrinsic, boundingbox, inimg->noDims, noBins);
}

void LibISR::Engine::ISRLowlevelEngine_GPU::computepfImageFromHistogram(UChar4Image *rgb_in, Objects::ISRHistogram *histogram)
{
	
	int w = rgb_in->noDims.width;
	int h = rgb_in->noDims.height;

	int noBins = histogram->noBins;

	Vector4u *inimg_ptr = rgb_in->GetData(MEMORYDEVICE_CUDA);
	float* histogram_ptr = histogram->getPosteriorHistogram(true);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)w / (float)blockSize.x), (int)ceil((float)h / (float)blockSize.y));

	computepfImageFromHistogram_device << <gridSize, blockSize >> >(inimg_ptr, histogram_ptr, rgb_in->noDims, noBins);
	rgb_in->UpdateHostFromDevice();
}


//////////////////////////////////////////////////////////////////////////
// device functions
//////////////////////////////////////////////////////////////////////////

__global__ void subsampleImageRGBDImage_device(const Vector4f *imageData_in, Vector2i oldDims, Vector4f *imageData_out, Vector2i newDims)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x > newDims.x - 1 || y > newDims.y - 1) return;

	filterSubsampleWithHoles(imageData_out, x, y, newDims, imageData_in, oldDims);
}

__global__ void prepareAlignedRGBDData_device(Vector4f* rgbd_out, const short *depth_in, const Vector4u *rgb_in, const Vector2i imgSize, Matrix3f H, Vector3f T)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x > imgSize.x - 1 || y > imgSize.y - 1) return;

	int idx = y * imgSize.x + x;
	ushort rawdepth = (ushort)depth_in[idx];
	float z = rawdepth == 65535 ? 0 : ((float)rawdepth / 1000.0f);

	// Vector3f uv_depth = (y, x, 1.0f);
	// Vector3f uv_color = z / 1000.0f * H * uv_depth + T / 1000.0f;

	// int X = static_cast<int>(uv_color[0] / uv_color[2]);
	// int Y = static_cast<int>(uv_color[1] / uv_color[2]);
	// int idx_depth_to_rgb = Y * 1920 + X;

	// if((X >= 0 && X < 1920) && (Y >= 0 && Y < 1080)){
	// 	rgbd_out[idx].x = rgb_in_ptr[idx_depth_to_rgb].r;
	// 	rgbd_out[idx].y = rgb_in_ptr[idx_depth_to_rgb].g;
	// 	rgbd_out[idx].z = rgb_in_ptr[idx_depth_to_rgb].b;
	// 	rgbd_out[idx].w = z;
	// }else{
	// 	rgbd_out[idx].x = 0;
	// 	rgbd_out[idx].y = 0;
	// 	rgbd_out[idx].z = 0;
	// 	rgbd_out[idx].w = z;
	// }


	if (T.x == 0 && T.y == 0 && T.z == 0)
	{
		rgbd_out[idx].x = rgb_in[idx].r;
		rgbd_out[idx].y = rgb_in[idx].g;
		rgbd_out[idx].z = rgb_in[idx].b;
		rgbd_out[idx].w = z;
		return;
	}
	rgbd_out[idx].w = z;
	mapRGBDtoRGB(rgbd_out[idx], Vector3f(x*z, y*z, z), rgb_in, imgSize, H, T);
}

__global__ void preparePointCloudFromAlignedRGBDImage_device(Vector4f* ptcloud_out, Vector4f* inimg, float* histogram, Vector4f intrinsic, Vector4i boundingbox, Vector2i imgSize, int histBins)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x > imgSize.x - 1 || y > imgSize.y - 1) return;
	
	int idx = y * imgSize.x + x;

	if (x < boundingbox.x || x >= boundingbox.z || y < boundingbox.y || y >= boundingbox.w)
	{ 
		ptcloud_out[idx] = Vector4f(0, 0, 0, -1);
	}
	else
	{
		float z = inimg[idx].w;
		unprojectPtWithIntrinsic(intrinsic, Vector3f(x*z, y*z, z), ptcloud_out[idx]);

		ptcloud_out[idx].w = getPf(inimg[idx], histogram, histBins);
	}
}

__global__ void computepfImageFromHistogram_device(Vector4u* inimg, float* histogram, Vector2i imgSize, int histBins)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x > imgSize.x - 1 || y > imgSize.y - 1) return;

	int idx = y * imgSize.x + x;
	float pf = getPf(inimg[idx], histogram, histBins);

	if (pf>0.5f)
	{
		inimg[idx].r = 255;
		inimg[idx].g = 0;
		inimg[idx].b = 0;
	}
	else if (pf==0.5f)
	{
		inimg[idx].r = 0;
		inimg[idx].g = 0;
		inimg[idx].b = 255;
	}

}

