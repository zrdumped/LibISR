#include "hip/hip_runtime.h"
#include "MatrixAssist_GPU.h"
#include <stdint.h>

using namespace LibISR::Engine;



__global__ void cudaTransform_device(uint8_t *output, uint8_t *input, uint32_t pitchOutput, uint32_t pitchInput, uint8_t bytesPerPixelOutput, uint8_t bytesPerPixelInput, float xRatio, float yRatio);


void LibISR::Engine::MatrixAssist::Test(){
    return;
}

//1920x1080 -> 640x480
void LibISR::Engine::MatrixAssist::Resize(const Vector4<unsigned char> *src, Vector4<unsigned char>* dst, 
				Vector2<int> srcSize, Vector2<int> dstSize)
{
    uint32_t src_row_btyes;
    uint32_t dst_row_bytes;
    int src_nb_component;
    int dst_nb_component;
    uint32_t src_size;
    uint32_t dst_size;
    uint8_t* device_src;
    uint8_t* device_dst;

    float x_ratio = dstSize.x / srcSize.x;
    float y_ratio = dstSize.y / srcSize.y;

    dim3 grid(dstSize.x, dstSize.y);
	dim3 blockSize(1, 1);

    src_row_btyes = (srcSize.x * 3 + 3) & ~3;
    dst_row_bytes = (dstSize.x * 4 + 3) & ~3;

    src_nb_component = 3;
    dst_nb_component = 4;
    src_size = srcSize.x * srcSize.y;
    dst_size = dstSize.x * dstSize.y;

    // Copy original image
    ORcudaSafeCall(hipMalloc((void **)&device_src, src_size));
    ORcudaSafeCall(hipMemcpy(device_src, src, src_size, hipMemcpyHostToDevice));
    ORcudaSafeCall(hipMalloc((void **)&device_dst, dst_size));
    cudaTransform_device << < grid, blockSize >> >(device_dst, device_src, dst_row_bytes, src_row_btyes, dst_nb_component, src_nb_component, x_ratio, y_ratio);
    
	// Copy scaled image to host
    ORcudaSafeCall(hipMemcpy(dst, device_dst, dst_size, hipMemcpyDeviceToHost));
    ORcudaSafeCall(hipFree(device_src));
    ORcudaSafeCall(hipFree(device_dst));
}

__global__ void cudaTransform_device(uint8_t *output, uint8_t *input, uint32_t pitchOutput, uint32_t pitchInput, uint8_t bytesPerPixelOutput, uint8_t bytesPerPixelInput, float xRatio, float yRatio)
{
	int x = (int)(xRatio * blockIdx.x);
	int y = (int)(yRatio * blockIdx.y);

	uint8_t *a; uint8_t *b; uint8_t *c; uint8_t *d;
	float xDist, yDist, blue, red, green;

	// X and Y distance difference
	xDist = (xRatio * blockIdx.x) - x;
	yDist = (yRatio * blockIdx.y) - y;

	// Points
	a = input + y * pitchInput + x * bytesPerPixelInput;
	b = input + y * pitchInput + (x + 1) * bytesPerPixelInput;
	c = input + (y + 1) * pitchInput + x * bytesPerPixelInput;
	d = input + (y + 1) * pitchInput + (x + 1) * bytesPerPixelInput;

	// blue
	blue = (a[2])*(1 - xDist)*(1 - yDist) + (b[2])*(xDist)*(1 - yDist) + (c[2])*(yDist)*(1 - xDist) + (d[2])*(xDist * yDist);

	// green
	green = ((a[1]))*(1 - xDist)*(1 - yDist) + (b[1])*(xDist)*(1 - yDist) + (c[1])*(yDist)*(1 - xDist) + (d[1])*(xDist * yDist);

	// red
	red = (a[0])*(1 - xDist)*(1 - yDist) + (b[0])*(xDist)*(1 - yDist) + (c[0])*(yDist)*(1 - xDist) + (d[0])*(xDist * yDist);

	uint8_t *p = output + blockIdx.y * pitchOutput + blockIdx.x * bytesPerPixelOutput;
	*(uint32_t*)p = 0xff000000 | ((((int)blue) << 16)) | ((((int)green) << 8)) | ((int)red);
}
